#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
// void Radix(int* array, int array_size, int max_digit); /* Thread function */
__host__ void rng(int* arr, int n); /* Seed function */
__host__ int max_el(int * vec, int n);
__host__ int num_digit(int el);
__device__ int to_digit(int el, int divider);
__host__ int to_digit_host(int el, int divider);
__host__ void print_array(int * array, int n);
__global__ void count_to_bucket(int * data, int * bucket, int length, int digit);
__host__ void countSort(int * data, int * bucket, int length, int digit);
__host__ void empty_bucket(int * bucket, int size);
__host__ void print_array_file(int * array, int array_len);

int main(int argc,char *argv[]) {
    if(argc != 2) {
        perror("Please specify data length");
        exit(1);
    }

    printf("flag 1\n");
    int data_size = strtol(argv[1], NULL, 10);
    int numThread = 1000;
    float numBlocksFloat = (float) data_size / numThread;
    int numBlocks = ceil(numBlocksFloat);
    int *global_array;
    int *global_bucket;
    int max_digit;
    int base= 10;
	 printf("data size : %d\n%.f\n", data_size,numBlocksFloat);
    printf("flag 2 thread %d block %d \n", numThread, numBlocks);
    // aloocating array to be accessible by both cpu and gpu
    hipMallocManaged(&global_array, data_size*sizeof(int)+1);
    // cudaMalloc(&local_array,data_size*sizeof(int)+1);
    rng(global_array, data_size);
    // cudaMemcpy(d_a, a, sizeof(float) * N, cudaMemcpyHostToDevice);
    printf("flag 3\n");
    printf("flag 4\n");
    // global_array = (*int )malloc(data_size * sizeof(int));
    // initialization data
    print_array(global_array, data_size);
    
    max_digit = num_digit(max_el(global_array, data_size));
    printf("max digit %d\n", max_digit);
    int bucket_el = base*max_digit;

    hipMallocManaged(&global_bucket, bucket_el*sizeof(int)+1);
    empty_bucket(global_bucket,bucket_el);
    for(int i = 1; i<= max_digit; i++){
	    count_to_bucket<<<numBlocks,numThread>>>(global_array,global_bucket,data_size,i);
    }
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    for(int i = 0; i<max_digit; i++){
        countSort(global_array, global_bucket, data_size, i);
    }
    print_array(global_bucket,bucket_el);
	print_array(global_array, data_size);
    hipFree(global_array);
    //cudaFree(global_bucket);
    return 0;
} 

__global__
void count_to_bucket(int * data, int * bucket, int length, int digit){
    int block = blockIdx.x;
    int thread = threadIdx.x;
    int i = block*1000+thread;
	// printf("block %d thread %d\n", digit, thread);
    //for(int i = (digit-1)*10 + thread;i <=(digit-1)*10+thread && i < length; i++){
    if(block*1000+thread < length){
		int num_bucket = to_digit(data[i], digit) + 10*(digit-1);
        printf("%d [%d] %d\n", data[i], digit,  num_bucket);
        bucket[num_bucket] ++;
	}
    //}
};


__host__
// void countSort(int * data, int * bucket, int length, int digit){
//     int *local_sort = (int*) malloc (length * sizeof(int));
//     int index = 0;

//     // sort
//     // printf("local sort ");
//     for(int block =0; block < digit; block++){
//         for(int d = 0; d < 10; d++){
//             for(int j = 0; j < length; j++){
//                 if(to_digit_host(data[j], block) == d){
//                     local_sort[index] = data[j];
//                     index ++;
//                     bucket[block*10+d] --;
//                 }
    
//                 if(bucket[block*10+d] == 0) {
//                     // printf("\n");
//                     break;
//                 }    
//             }
//         }    
//     }
//     // printf("index ends in %d \n", index);

//     // copy
//     for(int i=0; i < length; i++){
//         data[i] = local_sort[i];
//     }
//     free(local_sort);
//    //empty_bucket(bucket, 10);
// }

void countSort(int * data, int * bucket, int length, int digit){
    int * local_sort = (int*) malloc (length * sizeof(int));
    int index = 0;

    // sort
    // printf("local sort ");
    for(int i =0; i < 10; i++){
        for(int j = 0; j < length; j++){
            if(to_digit_host(data[j], digit+1) == i){
                local_sort[index] = data[j];
                index ++;
                bucket[digit*10+i] --;
            }

            if(bucket[digit*10+i] == 0) {
                // printf("\n");
                break;
            }
        }
    }
    // printf("index ends in %d \n", index);

    // copy
    for(int i=0; i < length; i++){
        data[i] = local_sort[i];
    }
    free(local_sort);
    empty_bucket(bucket, 10);
}
__host__
void empty_bucket(int * bucket, int size){
    for(int i = 0; i < size; i++){
        bucket[i] = 0;
    }
}

__host__
void rng(int* arr, int n) {
    int seed = 13516123;   
    srand(seed);
    for(long i = 0; i < n; i++) {
        arr[i] = (int)rand();
    }
}

__host__
int max_el(int * vec, int n){
    int max = vec[0];

    for(int i = 0; i < n; i++){
        if(vec[i] > max) max = vec[i];
    }

    return max;
};

__device__
int to_digit(int el, int divider){
    for(int i = 1; i< divider; i++){
        el /= 10;
    }
    return el % 10;
};

__host__ 
int to_digit_host(int el, int divider){
    for(int i = 1; i< divider; i++){
        el /= 10;
    }
    return el % 10;
};

__host__
void print_array(int * array, int array_len){
	int n = array_len;
    for(int i = 0; i < n; i++){
        printf("%d ", array[i]);
    }
    printf("\n");
}

__host__
void print_array_file(int * array, int array_len){
	int n = array_len;
    FILE * fp;
    FILE * fo;
    int i;
    /* open the file for writing*/
    fp = fopen ("../test/result.txt","w");
    fo = fopen ("../output/output.txt","w");
    /* write 10 lines of text into the file stream*/
    for(i = 0; i < n;i++){
       fprintf (fp, "%d ", array[i]);
       fprintf (fo, "%d ", array[i]);
    }
    fprintf (fp, "\n ");
    fprintf (fo, "\n ");

   /* close the file*/  
   fclose (fp);
   fclose (fo);
}

__host__
int num_digit(int el){
    int digit = 1;
    while(el > 9){
        el /= 10;
        digit++;
    }
    return digit;
};
