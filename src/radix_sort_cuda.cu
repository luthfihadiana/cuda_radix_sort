#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 10000000

__host__
int max_el(int * vec, int n){
    int max = vec[0];

    for(int i = 0; i < n; i++){
        if(vec[i] > max) max = vec[i];
    }

    return max;
};

__host__
void print_array(int * array, int array_len){
	int n = array_len;
    for(int i = 0; i < n; i++){
        printf("%d ", array[i]);
    }
    printf("\n");
}

__host__
void rng(int* arr, int n) {
    int seed = 13516123;   
    srand(seed);
    for(long i = 0; i < n; i++) {
        arr[i] = (int)rand();
    }
}

__host__
int max_digit(){
    return 0;
}

__global__
int to_digit(int el, int divider){
    for(int i = 1; i< divider; i++){
        el /= 10;
    }
    return el % 10;
};

int main(int argc,char *argv[]){
    if(argc != 2) {
        perror("Please specify data length");
        exit(1);
    }

    int data_size =  strtol(argv[1], NULL, 10);
    int * global_array;

    // aloocating array to be accessible by both cpu and gpu
    hipMallocManaged(&global_array, data_size*sizeof(int));
    
    rng(global_array, data_size);

    int max_digit = 

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    
    vector_add<<<1,1>>>(out, d_a, b, N);
    printf("%.f\n", out[0]);
    // Cleanup after kernel execution
    hipFree(d_a);
    free(a);
    
    return 0;
}