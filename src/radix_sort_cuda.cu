#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

__global__ void count_to_bucket(int * data, int * bucket, int length, int digit){
    for(int i = 0; i < length; i++){
        int num_bucket = to_digit(data[i], digit);
        // printf("%d [%d] %d\n", data[i], digit,  num_bucket);
        bucket[num_bucket] ++;
    }
};

__global__ void countSort(int * data, int * bucket, int length, int digit){
    int * local_sort = malloc (length * sizeof(int));
    int index = 0;

    // sort
    // printf("local sort ");
    for(int i =0; i < 10; i++){
        for(int j = 0; j < length; j++){
            if(to_digit(data[j], digit) == i){
                local_sort[index] = data[j];
                index ++;
                bucket[i] --;
            }

            if(bucket[i] == 0) {
                // printf("\n");
                break;
            }
        }
    }
    // printf("index ends in %d \n", index);

    // copy
    for(int i=0; i < length; i++){
        data[i] = local_sort[i];
    }
    free(local_sort);
    empty_bucket(bucket, 10);
}

__global__ void empty_bucket(int * bucket, int size){
    for(int i = 0; i < size; i++){
        bucket[i] = 0;
    }
}

__host__ void rng(int* arr, int n) {
    int seed = 13516123;   
    srand(seed);
    for(long i = 0; i < n; i++) {
        arr[i] = (int)rand();
    }
}

__device__ int max_el(int * vec, int n){
    int max = vec[0];

    for(int i = 0; i < n; i++){
        if(vec[i] > max) max = vec[i];
    }

    return max;
};

__device__ int to_digit(int el, int divider){
    for(int i = 1; i< divider; i++){
        el /= 10;
    }
    return el % 10;
};

__global__ void print_array(int * array, int array_len){
	int n = array_len;
    for(int i = 0; i < n; i++){
        printf("%d ", array[i]);
    }
    printf("\n");
}
__host__ void print_array_file(int * array, int array_len){
	int n = array_len;
    FILE * fp;
    FILE * fo;
    int i;
    /* open the file for writing*/
    fp = fopen ("../test/result.txt","w");
    fo = fopen ("../output/output.txt","w");
    /* write 10 lines of text into the file stream*/
    for(i = 0; i < n;i++){
       fprintf (fp, "%d ", array[i]);
       fprintf (fo, "%d ", array[i]);
    }
    fprintf (fp, "\n ");
    fprintf (fo, "\n ");

   /* close the file*/  
   fclose (fp);
   fclose (fo);
}

int main(){
    float *a, *b, *out;
    float *d_a;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Allocate device memory for a
    hipMalloc((void**)&d_a, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    
    vector_add<<<1,1>>>(out, d_a, b, N);
    printf("%.f\n", out[0]);
    // Cleanup after kernel execution
    hipFree(d_a);
    free(a);
    
    return 0;
}